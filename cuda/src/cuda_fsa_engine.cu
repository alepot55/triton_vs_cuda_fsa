#include "cuda_fsa_engine.h"
#include "cuda_utils.h" // Include the new CUDA utilities
#include <vector>
#include <string>
#include <cstring> // Per memcpy, memset
#include <numeric> // Per std::accumulate
#include <stdexcept>
#include <iostream>
#include <cassert>
#include <map>            // Usato temporaneamente in preparazione
#include <hip/hip_runtime.h> // Assicurati sia incluso per gli eventi CUDA
#include <iomanip>        // Per std::fixed, std::setprecision
#include "../../common/include/benchmark_metrics.h" // Updated include path
#include <chrono> // Per misurazione tempo
#include <algorithm> // Per std::copy
#include <iterator> // Per std::back_inserter
#include <iostream> // Per std::cerr
#include <cassert> // Per assert
#include <stdexcept> // Per std::runtime_error
#include <cstring> // Per memset

//-------------------------------------------------
// Implementazione Namespace CUDAFSAEngine
//-------------------------------------------------
namespace CUDAFSAEngine
{

    // Funzione Helper per creare GPUDFA dalla FSA host
    static GPUDFA prepareGPUDFAStructure(const FSA &fsa)
    {
        GPUDFA gpu_dfa;
        if (fsa.num_states > MAX_STATES || fsa.num_alphabet_symbols > MAX_SYMBOLS)
        {
            throw std::runtime_error("FSA exceeds maximum defined states or symbols.");
        }

        gpu_dfa.num_states = fsa.num_states;
        gpu_dfa.num_symbols = fsa.num_alphabet_symbols;
        gpu_dfa.start_state = fsa.start_state;

        // 1. Crea la mappa dei simboli (Lookup Ottimizzato)
        std::fill(std::begin(gpu_dfa.symbol_map), std::end(gpu_dfa.symbol_map), -1);
        std::map<char, int> symbol_to_index; // Mappa temporanea host
        for (int i = 0; i < fsa.num_alphabet_symbols; ++i)
        {
            if (static_cast<size_t>(i) < fsa.alphabet.size()) // Cast i to size_t
            {
                unsigned char c = static_cast<unsigned char>(fsa.alphabet[i]);
                gpu_dfa.symbol_map[c] = i;
                symbol_to_index[fsa.alphabet[i]] = i; // Popola mappa host
            }
            else
            {
                throw std::runtime_error("Inconsistent FSA: num_alphabet_symbols > alphabet size");
            }
        }

        // 2. Copia la matrice di transizione appiattita
        std::vector<int> flat(fsa.num_states * fsa.num_alphabet_symbols, -1); // Inizializza a -1
        for (int i = 0; i < fsa.num_states; ++i)
        {
            if (static_cast<size_t>(i) < fsa.transition_function.size()) // Cast i to size_t
            {
                for (int j = 0; j < fsa.num_alphabet_symbols; ++j)
                {
                    if (static_cast<size_t>(j) < fsa.transition_function[i].size()) // Cast j to size_t
                    {
                        flat[i * fsa.num_alphabet_symbols + j] = fsa.transition_function[i][j];
                    }
                }
            }
        }
        assert(flat.size() == static_cast<size_t>(fsa.num_states * fsa.num_alphabet_symbols));
        memcpy(gpu_dfa.transition_table, flat.data(), flat.size() * sizeof(int));

        // 3. Imposta gli stati accettanti
        memset(gpu_dfa.accepting_states, 0, MAX_STATES * sizeof(bool));
        for (int state : fsa.accepting_states)
        {
            if (state >= 0 && state < fsa.num_states)
            {
                gpu_dfa.accepting_states[state] = true;
            }
            else if (state >= MAX_STATES)
            {
                std::cerr << "Warning: Accepting state " << state << " exceeds MAX_STATES." << std::endl;
            }
        }
        return gpu_dfa;
    }

    // --- Implementazione Metodi Classe CUDAFSMRunner ---

    CUDAFSMRunner::CUDAFSMRunner(const FSA &fsa) : constant_memory_initialized(false) // Initialize members
    {
        initNVML(); // Initialize NVML when a runner is created

        h_dfa = prepareGPUDFAStructure(fsa); // Prepara la struttura host, inclusa la symbol_map

        // Alloca memoria globale per la tecnica GLOBAL_MEMORY
        CUDA_CHECK(hipMalloc(&d_dfa_global, sizeof(GPUDFA)));
        CUDA_CHECK(hipMemcpy(d_dfa_global, &h_dfa, sizeof(GPUDFA), hipMemcpyHostToDevice));
    }

    CUDAFSMRunner::~CUDAFSMRunner()
    {
        shutdownNVML(); // Shutdown NVML when runner is destroyed

        if (d_dfa_global)
        {
            hipError_t err = hipFree(d_dfa_global);
            if (err != hipSuccess)
            {
                fprintf(stderr, "CUDA error in ~CUDAFSMRunner (hipFree d_dfa_global): %s (%d)\n", hipGetErrorString(err), err);
            }
            d_dfa_global = nullptr;
        }
        freeGPUBuffers(); // Chiama la versione sicura
    }

    void CUDAFSMRunner::allocateGPUBuffers(size_t num_inputs, size_t total_input_chars)
    {
        if (num_inputs > allocated_num_inputs || total_input_chars > allocated_total_chars)
        {
            freeGPUBuffers();

            CUDA_CHECK(hipMalloc(&d_input_strings, total_input_chars * sizeof(char)));
            CUDA_CHECK(hipMalloc(&d_string_lengths, num_inputs * sizeof(int)));
            CUDA_CHECK(hipMalloc(&d_string_offsets, num_inputs * sizeof(int)));
            CUDA_CHECK(hipMalloc(&d_results, num_inputs * sizeof(char)));

            allocated_num_inputs = num_inputs;
            allocated_total_chars = total_input_chars;
        }
        else if (num_inputs == 0)
        {
            freeGPUBuffers();
        }
    }

    void CUDAFSMRunner::freeGPUBuffers()
    {
        hipError_t err;
        if (d_input_strings)
        {
            err = hipFree(d_input_strings);
            if (err != hipSuccess)
            {
                fprintf(stderr, "CUDA error in freeGPUBuffers (hipFree d_input_strings): %s (%d)\n", hipGetErrorString(err), err);
            }
            d_input_strings = nullptr;
        }
        if (d_string_lengths)
        {
            err = hipFree(d_string_lengths);
            if (err != hipSuccess)
            {
                fprintf(stderr, "CUDA error in freeGPUBuffers (hipFree d_string_lengths): %s (%d)\n", hipGetErrorString(err), err);
            }
            d_string_lengths = nullptr;
        }
        if (d_string_offsets)
        {
            err = hipFree(d_string_offsets);
            if (err != hipSuccess)
            {
                fprintf(stderr, "CUDA error in freeGPUBuffers (hipFree d_string_offsets): %s (%d)\n", hipGetErrorString(err), err);
            }
            d_string_offsets = nullptr;
        }
        if (d_results)
        {
            err = hipFree(d_results);
            if (err != hipSuccess)
            {
                fprintf(stderr, "CUDA error in freeGPUBuffers (hipFree d_results): %s (%d)\n", hipGetErrorString(err), err);
            }
            d_results = nullptr;
        }
        allocated_num_inputs = 0;
        allocated_total_chars = 0;
    }

    void CUDAFSMRunner::copyInputsToGPU(const std::vector<std::string>& inputs, std::vector<int>& h_lengths, std::vector<int>& h_offsets, std::vector<char>& h_concat_strings)
    {
        if (inputs.empty()) {
            allocateGPUBuffers(0, 0);
            return;
        }

        size_t num_inputs = inputs.size();
        h_lengths.resize(num_inputs);
        h_offsets.resize(num_inputs);
        h_concat_strings.clear();
        size_t estimated_total_chars = 0;
        for(const auto& s : inputs) estimated_total_chars += s.length();
        h_concat_strings.reserve(estimated_total_chars);

        int current_offset = 0;
        for (size_t i = 0; i < num_inputs; ++i) {
            const std::string& str = inputs[i];
            h_lengths[i] = static_cast<int>(str.length());
            h_offsets[i] = current_offset;
            h_concat_strings.insert(h_concat_strings.end(), str.begin(), str.end());
            current_offset += static_cast<int>(str.length());
        }
        size_t total_chars = h_concat_strings.size();

        allocateGPUBuffers(num_inputs, total_chars);

        if (num_inputs > 0) {
            CUDA_CHECK(hipMemcpy(d_input_strings, h_concat_strings.data(), total_chars * sizeof(char), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(d_string_lengths, h_lengths.data(), num_inputs * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(d_string_offsets, h_offsets.data(), num_inputs * sizeof(int), hipMemcpyHostToDevice));
        }
    }

    void CUDAFSMRunner::copyResultsFromGPU(std::vector<char> &gpu_results, size_t num_inputs)
    {
        if (num_inputs > 0 && d_results)
        {
            gpu_results.resize(num_inputs);
            CUDA_CHECK(hipMemcpy(gpu_results.data(), d_results, num_inputs * sizeof(char), hipMemcpyDeviceToHost));
        }
        else
        {
            gpu_results.clear();
        }
    }

    std::vector<bool> CUDAFSMRunner::runBatch(const std::vector<std::string> &inputs, CUDATechnique technique)
    {
        last_metrics = BenchmarkMetrics(); // Reset metrics
        auto start_total_time = std::chrono::high_resolution_clock::now();

        int num_strings = static_cast<int>(inputs.size());
        if (num_strings == 0)
        {
            return std::vector<bool>();
        }

        auto start_mem_input_time = std::chrono::high_resolution_clock::now();
        std::vector<int> h_string_lengths;
        std::vector<int> h_string_offsets;
        std::vector<char> h_input_strings;
        copyInputsToGPU(inputs, h_string_lengths, h_string_offsets, h_input_strings);
        CUDA_CHECK(hipDeviceSynchronize());
        auto end_mem_input_time = std::chrono::high_resolution_clock::now();
        last_metrics.memory_transfer_time_ms += std::chrono::duration<double, std::milli>(end_mem_input_time - start_mem_input_time).count();

        int block_size = BLOCK_SIZE;
        int grid_size = (num_strings + block_size - 1) / block_size;

        hipEvent_t start_event, stop_event;
        CUDA_CHECK(hipEventCreate(&start_event));
        CUDA_CHECK(hipEventCreate(&stop_event));

        double start_gpu_util = getGPUUtilization();

        CUDA_CHECK(hipEventRecord(start_event));

        switch (technique)
        {
        case CUDATechnique::GLOBAL_MEMORY:
            if (!d_dfa_global) throw std::runtime_error("DFA global memory not initialized.");
            fsa_kernel_global<<<grid_size, block_size>>>(d_dfa_global, d_input_strings,
                                                         d_string_lengths, d_string_offsets,
                                                         num_strings, d_results);
            break;

        case CUDATechnique::CONSTANT_MEMORY:
#ifdef __HIPCC__
            if (!constant_memory_initialized) {
                 CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_dfa), &h_dfa, sizeof(GPUDFA)));
                 constant_memory_initialized = true;
            }
            fsa_kernel_constant<<<grid_size, block_size>>>(d_input_strings,
                                                           d_string_lengths, d_string_offsets,
                                                           num_strings, d_results);
#else
            CUDA_CHECK(hipEventDestroy(start_event));
            CUDA_CHECK(hipEventDestroy(stop_event));
            throw std::runtime_error("Constant memory kernel requires compilation with NVCC.");
#endif
            break;

        case CUDATechnique::SHARED_MEMORY:
#ifdef __HIPCC__
             if (!d_dfa_global) throw std::runtime_error("DFA global memory not initialized (needed for shared mem load).");
             fsa_kernel_shared<<<grid_size, block_size>>>(d_dfa_global, d_input_strings,
                                                          d_string_lengths, d_string_offsets,
                                                          num_strings, d_results);
#else
            CUDA_CHECK(hipEventDestroy(start_event));
            CUDA_CHECK(hipEventDestroy(stop_event));
            throw std::runtime_error("Shared memory kernel requires compilation with NVCC.");
#endif
            break;

        default:
            CUDA_CHECK(hipEventDestroy(start_event));
            CUDA_CHECK(hipEventDestroy(stop_event));
            throw std::runtime_error("Invalid CUDA technique specified.");
        }

        CUDA_CHECK(hipEventRecord(stop_event));
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipEventSynchronize(stop_event));
        float kernel_time = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&kernel_time, start_event, stop_event));
        last_metrics.kernel_time_ms = kernel_time;

        double end_gpu_util = getGPUUtilization();
        if (start_gpu_util >= 0 && end_gpu_util >= 0) {
             last_metrics.gpu_utilization_percent = static_cast<float>((start_gpu_util + end_gpu_util) / 2.0);
        } else {
             last_metrics.gpu_utilization_percent = 0.0f;
        }

        CUDA_CHECK(hipEventDestroy(start_event));
        CUDA_CHECK(hipEventDestroy(stop_event));

        auto start_mem_output_time = std::chrono::high_resolution_clock::now();
        std::vector<char> gpu_results;
        copyResultsFromGPU(gpu_results, num_strings);
        CUDA_CHECK(hipDeviceSynchronize());
        auto end_mem_output_time = std::chrono::high_resolution_clock::now();
        last_metrics.memory_transfer_time_ms += std::chrono::duration<double, std::milli>(end_mem_output_time - start_mem_output_time).count();

        auto end_total_time = std::chrono::high_resolution_clock::now();
        last_metrics.execution_time_ms = std::chrono::duration<double, std::milli>(end_total_time - start_total_time).count();

        last_metrics.memory_used_bytes = getMemoryUsage();

        size_t input_data_size = h_input_strings.size() * sizeof(char) +
                                 h_string_lengths.size() * sizeof(int) +
                                 h_string_offsets.size() * sizeof(int);
        size_t output_data_size = num_strings * sizeof(char);
        size_t total_transfer_bytes = input_data_size + output_data_size;
        if (last_metrics.memory_transfer_time_ms > 0) {
            last_metrics.memory_bandwidth_MBps = (static_cast<float>(total_transfer_bytes) / (1024.0f * 1024.0f)) / (last_metrics.memory_transfer_time_ms / 1000.0f);
        } else {
            last_metrics.memory_bandwidth_MBps = 0.0f;
        }

        std::vector<bool> bool_results(num_strings);
        for (int i = 0; i < num_strings; ++i)
        {
            bool_results[i] = (gpu_results[i] != 0);
        }

        return bool_results;
    }

    BenchmarkMetrics CUDAFSMRunner::getLastMetrics() const {
        return last_metrics;
    }

} // namespace CUDAFSAEngine
