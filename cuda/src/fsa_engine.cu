#include "hip/hip_runtime.h"
#include "fsa_engine.h"
#include <stack>
#include <set>
#include <map>
#include <algorithm>
#include <iostream>
#include <sstream>
#include <queue>
#include <unordered_map>
#include <unordered_set>
#include <hip/hip_runtime.h>
#include <>

// ============ CUDA kernel implementations ============

// Kernel legacy per compatibilità con il benchmark esistente
__global__ void fsa_kernel(const CUDAFSA* fsa, const char* input_string, bool* output) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Recupera lo stato di partenza
    int current_state = fsa->start_state;
    
    // Elabora la stringa di input
    for (int i = 0; input_string[i] != '\0'; i++) {
        char c = input_string[i];
        int symbol = -1;
        
        // Match character to valid symbol index
        if (c == '0') symbol = 0;
        else if (c == '1') symbol = 1;
        else {
            // For now we only support binary inputs in the kernel
            // A full implementation would need a proper mapping here
            output[thread_id] = false;
            return;
        }
        
        // Verifica se esiste una transizione valida
        if (symbol >= fsa->num_alphabet_symbols) {
            output[thread_id] = false;
            return;
        }
        
        int next_state = fsa->transition_matrix[current_state * MAX_SYMBOLS + symbol];
        if (next_state < 0) {
            output[thread_id] = false;
            return;
        }
        
        current_state = next_state;
    }
    
    // Verifica se lo stato corrente è uno stato di accettazione
    bool accepts = false;
    for (int i = 0; i < fsa->num_accepting_states; i++) {
        if (current_state == fsa->accepting_states[i]) {
            accepts = true;
            break;
        }
    }
    
    output[thread_id] = accepts;
}

// Kernel ottimizzato per processare un batch di stringhe di lunghezza variabile
__global__ void fsa_kernel_batch(const GPUDFA* dfa, const char* input_strings, 
                           const int* string_lengths, const int* string_offsets,
                           int num_strings, char* results) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_strings) return;
    
    // Recupero informazioni sulla stringa
    int offset = string_offsets[tid];
    int length = string_lengths[tid];
    
    // Inizializzazione dello stato corrente
    int current_state = dfa->start_state;
    
    // Empty string case - check if start state is accepting
    if (length == 0) {
        results[tid] = dfa->accepting_states[current_state] ? 1 : 0;
        return;
    }
    
    // Memoria condivisa per caching delle transizioni frequenti
    __shared__ int transition_cache[BLOCK_SIZE][2]; // [thread_idx][state, symbol] -> next_state
    __shared__ int cache_hits[BLOCK_SIZE];
    cache_hits[threadIdx.x] = -1; // -1 indica cache vuota
    
    // Elaborazione dei caratteri della stringa di input
    for (int i = 0; i < length; i++) {
        char c = input_strings[offset + i];
        int symbol = -1;
        
        // Map character to symbol index
        if (c == '0') symbol = 0;
        else if (c == '1') symbol = 1;
        else {
            // Currently only support binary alphabet in GPU kernels
            results[tid] = 0;  // false
            return;
        }
        
        // Controlla nella cache se questa transizione è stata già usata
        if (cache_hits[threadIdx.x] >= 0 && 
            transition_cache[threadIdx.x][0] == current_state && 
            transition_cache[threadIdx.x][1] == symbol) {
            current_state = cache_hits[threadIdx.x];
        } else {
            // Altrimenti, cerca nella tabella di transizione
            int transition_idx = current_state * MAX_SYMBOLS + symbol;
            int next_state = dfa->transition_table[transition_idx];
            
            // Aggiorna la cache
            if (next_state != -1) {
                transition_cache[threadIdx.x][0] = current_state;
                transition_cache[threadIdx.x][1] = symbol;
                cache_hits[threadIdx.x] = next_state;
                current_state = next_state;
            } else {
                // Nessuna transizione valida, DFA rifiuta
                results[tid] = 0;  // false
                return;
            }
        }
    }
    
    // Controllo se lo stato finale è di accettazione
    results[tid] = dfa->accepting_states[current_state] ? 1 : 0;
}

// Kernel ottimizzato per stringhe di lunghezza fissa (meno divergenza dei thread)
__global__ void fsa_kernel_fixed_length(const GPUDFA* dfa, const char* input_strings, 
                                      int string_length, int num_strings, char* results) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_strings) return;
    
    // Empty string case - check if start state is accepting
    if (string_length == 0) {
        results[tid] = dfa->accepting_states[dfa->start_state] ? 1 : 0;
        return;
    }
    
    // Offset per questa stringa
    int offset = tid * string_length;
    
    // Stato iniziale
    int current_state = dfa->start_state;
    
    // Elaborazione della stringa
    for (int i = 0; i < string_length; i++) {
        char c = input_strings[offset + i];
        int symbol = -1;
        
        // Map character to symbol index
        if (c == '0') symbol = 0;
        else if (c == '1') symbol = 1;
        else {
            // Currently only support binary alphabet in GPU kernels
            results[tid] = 0;  // false
            return;
        }
        
        // Accesso coalescente alla memoria globale
        int next_state = dfa->transition_table[current_state * MAX_SYMBOLS + symbol];
        
        if (next_state == -1) {
            // Nessuna transizione valida, rifiuta
            results[tid] = 0;  // false
            return;
        }
        
        current_state = next_state;
    }
    
    // Controlla se lo stato finale è uno stato di accettazione
    results[tid] = dfa->accepting_states[current_state] ? 1 : 0;
}

// ============ Host code for GPU execution ============

// Funzione di utilità per convertire FSA in CUDAFSA
CUDAFSA convertToCUDAFSA(const FSA& fsa) {
    std::cout << "Converting FSA to CUDAFSA" << std::endl;
    
    CUDAFSA cuda_fsa;
    cuda_fsa.num_states = fsa.num_states;
    cuda_fsa.num_alphabet_symbols = fsa.num_alphabet_symbols;
    cuda_fsa.start_state = fsa.start_state;
    cuda_fsa.num_accepting_states = fsa.accepting_states.size();
    
    // Inizializza la tabella con -1 (nessuna transizione)
    for (int i = 0; i < MAX_STATES * MAX_SYMBOLS; i++) {
        cuda_fsa.transition_matrix[i] = -1;
    }
    
    // Copia la tabella di transizione
    for (int state = 0; state < fsa.num_states && state < MAX_STATES; state++) {
        for (int symbol = 0; symbol < fsa.num_alphabet_symbols && symbol < MAX_SYMBOLS; symbol++) {
            if (state < fsa.transition_function.size() && 
                symbol < fsa.transition_function[state].size() &&
                fsa.transition_function[state][symbol] >= 0) {
                cuda_fsa.transition_matrix[state * MAX_SYMBOLS + symbol] = 
                    fsa.transition_function[state][symbol];
            }
        }
    }
    
    // Copia gli stati di accettazione
    int i = 0;
    for (int state : fsa.accepting_states) {
        if (i < MAX_STATES) {
            cuda_fsa.accepting_states[i++] = state;
        }
    }
    
    std::cout << "CUDAFSA conversion completed: " << cuda_fsa.num_states << " states, " 
              << cuda_fsa.num_accepting_states << " accepting states" << std::endl;
    
    return cuda_fsa;
}

// Prepara il DFA per l'esecuzione GPU
GPUDFA FSAEngine::prepareGPUDFA(const FSA& fsa) {
    std::cout << "Preparing GPU DFA" << std::endl;
    
    GPUDFA gpu_dfa;
    gpu_dfa.num_states = fsa.num_states;
    gpu_dfa.num_symbols = fsa.num_alphabet_symbols;
    gpu_dfa.start_state = fsa.start_state;
    
    // Inizializza tabella delle transizioni e stati di accettazione
    memset(gpu_dfa.transition_table, -1, sizeof(gpu_dfa.transition_table)); // -1 indica nessuna transizione
    memset(gpu_dfa.accepting_states, 0, sizeof(gpu_dfa.accepting_states));
    
    // Copia la tabella delle transizioni in formato linearizzato per accesso rapido
    for (int state = 0; state < fsa.num_states; state++) {
        for (int symbol = 0; symbol < fsa.num_alphabet_symbols; symbol++) {
            // Verifica se esiste una transizione valida
            if (state < fsa.transition_function.size() && 
                symbol < fsa.transition_function[state].size() &&
                fsa.transition_function[state][symbol] >= 0) {
                gpu_dfa.transition_table[state * MAX_SYMBOLS + symbol] = 
                    fsa.transition_function[state][symbol];
            }
        }
    }
    
    // Imposta gli stati di accettazione
    for (int state : fsa.accepting_states) {
        if (state < MAX_STATES) {
            gpu_dfa.accepting_states[state] = true;
        }
    }
    
    std::cout << "GPU DFA prepared successfully" << std::endl;
    return gpu_dfa;
}

// Implementazione del metodo per eseguire un batch di stringhe sulla GPU
std::vector<bool> FSAEngine::runBatchOnGPU(const FSA& fsa, const std::vector<std::string>& inputs) {
    if (inputs.empty()) {
        return {}; // Return empty vector for empty input
    }
    
    try {
        // Prepare the GPU DFA
        GPUDFA gpu_dfa = prepareGPUDFA(fsa);
        
        // Calculate total size needed for all strings
        size_t total_chars = 0;
        for (const auto& s : inputs) {
            total_chars += s.length();
        }
        
        // Prepare host data
        std::vector<char> all_strings(total_chars);
        std::vector<int> string_lengths(inputs.size());
        std::vector<int> string_offsets(inputs.size());
        
        // Fill input data
        size_t offset = 0;
        for (size_t i = 0; i < inputs.size(); i++) {
            string_offsets[i] = offset;
            string_lengths[i] = inputs[i].length();
            
            if (!inputs[i].empty()) {
                std::copy(inputs[i].begin(), inputs[i].end(), all_strings.begin() + offset);
                offset += inputs[i].length();
            }
        }
        
        // Allocate device memory
        char* d_strings;
        int* d_lengths;
        int* d_offsets;
        char* d_results;
        GPUDFA* d_dfa;
        
        hipMalloc(&d_strings, all_strings.size());
        hipMalloc(&d_lengths, string_lengths.size() * sizeof(int));
        hipMalloc(&d_offsets, string_offsets.size() * sizeof(int));
        hipMalloc(&d_results, inputs.size() * sizeof(char));
        hipMalloc(&d_dfa, sizeof(GPUDFA));
        
        // Copy data to device
        hipMemcpy(d_strings, all_strings.data(), all_strings.size(), hipMemcpyHostToDevice);
        hipMemcpy(d_lengths, string_lengths.data(), string_lengths.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_offsets, string_offsets.data(), string_offsets.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dfa, &gpu_dfa, sizeof(GPUDFA), hipMemcpyHostToDevice);
        
        // Launch kernel
        int block_size = BLOCK_SIZE;
        int grid_size = (inputs.size() + block_size - 1) / block_size;
        
        // Check if all strings are the same length for optimization
        bool same_length = true;
        int first_length = inputs[0].length();
        for (size_t i = 1; i < inputs.size(); i++) {
            if (inputs[i].length() != static_cast<size_t>(first_length)) {
                same_length = false;
                break;
            }
        }
        
        // Choose the appropriate kernel based on input characteristics
        if (same_length) {
            // Use the fixed length kernel for better performance
            fsa_kernel_fixed_length<<<grid_size, block_size>>>(d_dfa, d_strings, first_length, inputs.size(), d_results);
        } else {
            // Use the variable length kernel
            fsa_kernel_batch<<<grid_size, block_size>>>(d_dfa, d_strings, d_lengths, d_offsets, inputs.size(), d_results);
        }
        
        // Synchronize to ensure completion
        hipDeviceSynchronize();
        
        // Check for errors
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
            throw std::runtime_error("CUDA execution failed");
        }
        
        // Retrieve results
        std::vector<char> results(inputs.size());
        hipMemcpy(results.data(), d_results, inputs.size() * sizeof(char), hipMemcpyDeviceToHost);
        
        // Convert to boolean vector
        std::vector<bool> bool_results(inputs.size());
        for (size_t i = 0; i < inputs.size(); i++) {
            bool_results[i] = (results[i] != 0);
        }
        
        // Free device memory
        hipFree(d_strings);
        hipFree(d_lengths);
        hipFree(d_offsets);
        hipFree(d_results);
        hipFree(d_dfa);
        
        return bool_results;
    } catch (const std::exception& e) {
        std::cerr << "Error in runBatchOnGPU: " << e.what() << std::endl;
        // Return empty vector on error
        return std::vector<bool>(inputs.size(), false);
    }
}
