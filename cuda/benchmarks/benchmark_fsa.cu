#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>
#include <nvml.h>

#include "../../common/benchmark/benchmark_metrics.h"
#include "../../common/test/test_case.h"
#include "../../common/benchmark/cmdline.h"
#include "../include/fsa_engine.h"
#include "../../common/include/fsa_definition.h"

// External declarations for NVML functions
extern bool initNVML();
extern void shutdownNVML();

int main(int argc, char* argv[]) {
    try {
        // Default values
        std::string regex;
        std::string input;
        int batch_size;
        bool verbose;
        std::string test_file;
        bool run_tests = false;
        
        // Parse command line arguments
        parseArgs(argc, argv, regex, input, batch_size, verbose, test_file, run_tests);
        
        // If help was requested, exit
        if (std::string(argc > 1 ? argv[1] : "") == "--help") {
            return 0;
        }
        
        // Initialize NVML
        if (!initNVML()) {
            std::cerr << "Could not initialize NVML. Some metrics will be unavailable." << std::endl;
        }
        
        // Run tests if requested
        if (run_tests) {
            std::vector<TestCase> tests;
            if (loadTestsFromFile(test_file, tests)) {
                runAllTests(tests, batch_size, verbose);
                shutdownNVML();
                return 0;
            } else {
                shutdownNVML();
                return 1;
            }
        }
        
        // Print benchmark info
        if (verbose) {
            std::cout << "Regex: " << regex << std::endl;
            std::cout << "Testing string: " << input << std::endl;
            std::cout << "Batch size: " << batch_size << std::endl;
            std::cout << "Mode: GPU-optimized CUDA (default)" << std::endl;
        }
        
        // Convert regex to FSA
        FSA fsa;
        if (verbose) {
            std::cout << "Step 1: Converting regex to FSA..." << std::endl;
            fsa = FSAEngine::regexToDFA(regex);
            std::cout << "FSA created with " << fsa.num_states << " states" << std::endl;
        } else {
            // Suppress output
            std::streambuf* old_cout = std::cout.rdbuf();
            std::ofstream null_stream;
            null_stream.open("/dev/null");
            std::cout.rdbuf(null_stream.rdbuf());
            
            fsa = FSAEngine::regexToDFA(regex);
            
            // Restore output
            std::cout.rdbuf(old_cout);
        }
        
        // Create batch of input strings
        std::vector<std::string> inputs(batch_size, input);
        std::vector<bool> results;
        
        // Process inputs in batch mode
        if (verbose) std::cout << "Step 2: Running FSA on GPU..." << std::endl;
        auto start_time = std::chrono::high_resolution_clock::now();
        results = FSAEngine::runBatchOnGPU(fsa, inputs);
        auto end_time = std::chrono::high_resolution_clock::now();
        
        // Calculate execution time
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
        double execution_time_ms = duration.count() / 1000.0;
        
        // Output results
        if (!results.empty()) {
            std::cout << "Benchmark: CUDA" << std::endl;
            std::cout << "Input String: " << input << std::endl;
            std::cout << "Accepts: " << (results[0] ? "true" : "false") << std::endl;
            std::cout << "Execution Time (total): " << std::fixed << std::setprecision(3) 
                      << execution_time_ms << " ms" << std::endl;
        } else {
            std::cerr << "Error: No results returned" << std::endl;
        }
        
        // Shutdown NVML
        shutdownNVML();
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "Exception caught: " << e.what() << std::endl;
        shutdownNVML();
        return 1;
    } catch (...) {
        std::cerr << "Unknown exception caught" << std::endl;
        shutdownNVML();
        return 1;
    }
}