#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include "fsa_engine.h"
#include "fsa_definition.h"

int main() {
    // Esempio di FSA (placeholder)
    FSA fsa;
    fsa.num_states = 2;
    fsa.num_alphabet_symbols = 2;
    fsa.transition_function = {{1, 0}, {1, 1}}; // Esempio di funzione di transizione
    fsa.start_state = 0;
    fsa.accepting_states = {1};

    std::string input_string = "0101";

    FSA* dev_fsa;
    char* dev_input_string;
    bool* dev_output;
    bool host_output;

    // Allocazione memoria su device
    hipMalloc(&dev_fsa, sizeof(FSA));
    hipMalloc(&dev_input_string, input_string.length() + 1); // +1 per il terminatore null
    hipMalloc(&dev_output, sizeof(bool));

    // Copia dati host -> device
    hipMemcpy(dev_fsa, &fsa, sizeof(FSA), hipMemcpyHostToDevice);
    hipMemcpy(dev_input_string, input_string.c_str(), input_string.length() + 1, hipMemcpyHostToDevice);

    // Esecuzione kernel
    dim3 blockDim(256); // Esempio blockDim
    dim3 gridDim(1);    // Esempio gridDim
    fsa_kernel<<<gridDim, blockDim>>>(dev_fsa, dev_input_string, dev_output);

    // Copia risultato device -> host
    hipMemcpy(&host_output, dev_output, sizeof(bool), hipMemcpyDeviceToHost);

    std::cout << "Input string: " << input_string << std::endl;
    std::cout << "FSA accepts: " << (host_output ? "true" : "false") << std::endl;

    // Free memory device
    hipFree(dev_fsa);
    hipFree(dev_input_string);
    hipFree(dev_output);

    return 0;
}