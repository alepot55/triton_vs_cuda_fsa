#include <iostream>
#include <vector>
#include <string>
#include <chrono> // Per misurare il tempo
#include <hip/hip_runtime.h>
#include "fsa_engine.h"
#include "fsa_definition.h"

int main() {
    // Esempio di FSA (placeholder)
    FSA fsa;
    fsa.num_states = 2;
    fsa.num_alphabet_symbols = 2;
    fsa.transition_function = {{1, 0}, {1, 1}}; // Esempio di funzione di transizione
    fsa.start_state = 0;
    fsa.accepting_states = {1};

    std::string input_string = "0101";

    FSA* dev_fsa;
    char* dev_input_string;
    bool* dev_output;
    bool host_output;

    // Allocazione memoria su device
    hipMalloc(&dev_fsa, sizeof(FSA));
    hipMalloc(&dev_input_string, input_string.length() + 1); // +1 per il terminatore null
    hipMalloc(&dev_output, sizeof(bool));

    // Copia dati host -> device
    hipMemcpy(dev_fsa, &fsa, sizeof(FSA), hipMemcpyHostToDevice);
    hipMemcpy(dev_input_string, input_string.c_str(), input_string.length() + 1, hipMemcpyHostToDevice);

    auto start_time = std::chrono::high_resolution_clock::now(); // Inizio misurazione tempo

    // Esecuzione kernel
    dim3 blockDim(256);
    dim3 gridDim(1);
    fsa_kernel<<<gridDim, blockDim>>>(dev_fsa, dev_input_string, dev_output);

    hipDeviceSynchronize(); // Importante sincronizzare per misurare il tempo correttamente

    auto end_time = std::chrono::high_resolution_clock::now(); // Fine misurazione tempo
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    double execution_time_ms = duration.count() / 1000.0; // Tempo in millisecondi

    // Copia risultato device -> host
    hipMemcpy(&host_output, dev_output, sizeof(bool), hipMemcpyDeviceToHost);

    // Output strutturato per parsing
    std::cout << "Benchmark: CUDA" << std::endl;
    std::cout << "Input String: " << input_string << std::endl;
    std::cout << "Accepts: " << (host_output ? "true" : "false") << std::endl;
    std::cout << "Execution Time (ms): " << execution_time_ms << std::endl;

    // Free memory device
    hipFree(dev_fsa);
    hipFree(dev_input_string);
    hipFree(dev_output);

    return 0;
}